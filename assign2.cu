#include "hip/hip_runtime.h"
//CSCI415 - Assignment 2
//Original by: Saeed Salem, 2/25/2015
//Updated by: Otto Borchert, 2/20/2017
//To compile: make clean; make
//To run: ./assign2

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <math.h>
#include <iomanip>
#include <string>
#include <sys/time.h>

typedef std::vector< std::vector<int> > AdjacencyMatrix;
AdjacencyMatrix adjMatrix;

int threads_per_block = 256;
int blocks = (int)(10000/threads_per_block+1);
int n;

//void printAdjMatrix(AdjacencyMatrix adjMatrix)
//{
    //for (int i=0; i<adjMatrix.size(); i++)
    //{
        //for (int j=0; j<adjMatrix[i].size(); j++) 
        //{
      //      std::cout << adjMatrix[i][j] << " ";
    //    }
  //      std::cout << std::endl;
//    }
//}
__global__ void clustCoeff_Parallel(int *matrix, float *output,int n)
{
    double totalC = 0.0;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < n)
    {
	const int size = (n*n);
	
        int *temp = new int[size];
        int nCount = 0;
        int mCount = 0;
        for(int y = 0;y<n;y++)
        {
	    int b = matrix[x*n+y];
            if(b==1)
            {
                temp[nCount]=y;
                nCount++;
            }

        }
       
        for(int p =0;p<nCount;p++ )
        {
            for(int q =0;q<n;q++)
            {
                if(matrix[temp[p]*n+q] == 1 && matrix[q*n+x] == 1)
                {
                    mCount++;	    
                }
            }
                
        }
        
        output[x]=((mCount)/(nCount*(nCount-1.0)));
        totalC += output[x];
   }
}
double clustCoeff_Serial(AdjacencyMatrix matrix)
{  std::vector<double> total;
        double totalC = 0.0;
        for(int x =0;x<n;x++)
        {
            //Parallelize this hunk
            std::vector<int> temp;
            int nCount = 0;
            int mCount = 0;
            for(int y = 0;y<n;y++)
            {
                if(matrix[x][y])
                {
                    temp.push_back(y);
                    nCount++;
                }

            }
            for(int p =0;p<temp.size();p++ )
            {
                for(int q =0;q<n;q++)
                {
                    if(matrix[temp[p]][q] && matrix[q][x])
                    {
                        mCount++;
                    }
                }
                
            }
            //std::cout<<mCount<<std::endl;
            total.push_back((mCount)/(nCount*(nCount-1.0)));
            std::cout<<x<<": "<<total[x]<<std::endl;
            totalC += total[x];
        }
        std::cout<<totalC<<std::endl;
        double result = ((1.0/n)*totalC);
        std::cout<<"Total: "<<result<<std::endl;
        return 0.0;//result;
}
void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}
int main()
{
    std::fstream myfile("toyGraph1.txt",std::ios_base::in);
    int u,v;
    int maxNode = 0;
    std::vector< std::pair<int,int> > allEdges;
    while(myfile >> u >> v)
    {
        allEdges.push_back(std::make_pair(u,v));
        if(u > maxNode)
          maxNode = u;

        if(v > maxNode)
          maxNode = v;                 
    }

    n = maxNode + 1;  //Since nodes starts with 0
    std::cout << "Graph has " << n << " nodes" << std::endl;

    adjMatrix = AdjacencyMatrix(n,std::vector<int>(n,0));
    //populate the matrix
    for(int i =0; i<allEdges.size() ; i++){
       u = allEdges[i].first;
       v = allEdges[i].second;
       adjMatrix[u][v] = 1;
       adjMatrix[v][u] = 1;
    } 
    //You can also make a list of neighbors for each node if you want.
    //printAdjMatrix(adjMatrix);


    //TODO: Write serial clustering coefficent code; include timing and error checking
    std::cout<<"Serial computation:"<<std::endl;
    clustCoeff_Serial(adjMatrix);
    std::cout<<std::endl<<"Parallel computation:"<<std::endl;
    //TODO: Write parallel clustering cofficient code; include timing and error checking
    int *d_input;
    int *d_temp = new int[n*n];
    float *d_output;
    float *h_gpu_result = (float*)malloc(n*sizeof(float));
    hipMalloc((void **) &d_input, sizeof(int)*n*n);
    hipMalloc((void **) &d_output, n*sizeof(float));
    checkErrors("MAlloc");
    for (int i =0;i<n;i++){
	for(int j =0;j<n;j++){
	d_temp[(i*n)+j]=adjMatrix[i][j]; 
	}
    }
    //Source array maps values properly
    //Copying source array to device seems to not be working
    hipMemcpy(d_input, d_temp, (n*n*sizeof(int)), hipMemcpyHostToDevice);
    checkErrors("memCopy");

    clustCoeff_Parallel<<<blocks,threads_per_block>>>(d_input,d_output,n);
    //hipDeviceSynchronize();
    hipMemcpy(h_gpu_result, d_output, n*sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    //hipFree(d_output);
    //hipFree(d_input);
    float coef=0.0;
    for(int j =0;j<n;j++){
    coef += h_gpu_result[j];
    std::cout<<j<<": "<<h_gpu_result[j]<<std::endl;
    }
        
    std::cout<<"Coeffecient is: "<<(coef/n)<<std::endl;
    //TODO: Compare serial and parallel results

    return 0;
}

