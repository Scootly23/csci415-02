#include "hip/hip_runtime.h"
//CSCI415 - Assignment 2
//Original by: Saeed Salem, 2/25/2015
//Updated by: Otto Borchert, 2/20/2017
//To compile: make clean; make
//To run: ./assign2

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <math.h>
#include <iomanip>
#include <string>
#include <sys/time.h>

typedef std::vector< std::vector<int> > AdjacencyMatrix;
AdjacencyMatrix adjMatrix;

int threads_per_block = 256;
int blocks = (int)(10000/threads_per_block+1);
int n;

void printAdjMatrix(AdjacencyMatrix adjMatrix)
{
    for (int i=0; i<adjMatrix.size(); i++)
    {
        for (int j=0; j<adjMatrix[i].size(); j++) 
        {
            std::cout << adjMatrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
}
__global__ void clustCoeff_Parallel(float *matrix, float *output,int n)
{
    double totalC = 0.0;
    int x = blockDim.x * blockIdx.x + threadIdx.x;
    if(x < n)
    {
	int size = (n*n);
	printf("%d\n",size);
	
        int *temp = new int[size];
        int nCount = 0;
        int mCount = 0;
        for(int y = 0;y<n;y++)
        { 
	    int b = matrix[x*n+y];
            if(b==1)
            {
                temp[nCount]=y;
                nCount++;
            }

        }
        //This only does 6 iterations even though size is set to 36
	for(int i=0;i<size;i++)
	{
        //Values from source array not translating properly
	printf("m: %d\n",matrix[i]);
	}
        for(int p =0;p<nCount;p++ )
        {
            for(int q =0;q<n;q++)
            {
		printf("%d\n",(temp[p]*n+q));
                if(matrix[temp[p]*n+q] == 1 && matrix[q*n+x] == 1)
                {
                    //mCount++;
		    
		    
                }
            }
                
        }
        
        //output[x]=((mCount)/(nCount*(nCount-1.0)));
        //totalC += output[x];
   }
        //output[x] = 4;

}
double clustCoeff_Serial(AdjacencyMatrix matrix)
{  std::vector<double> total;
        double totalC = 0.0;
        for(int x =0;x<n;x++)
        {
            //Parallelize this hunk
            std::vector<int> temp;
            int nCount = 0;
            int mCount = 0;
            for(int y = 0;y<n;y++)
            {
                if(matrix[x][y])
                {
                    temp.push_back(y);
                    nCount++;
                }

            }
            for(int p =0;p<temp.size();p++ )
            {
                for(int q =0;q<n;q++)
                {
                    if(matrix[temp[p]][q] && matrix[q][x])
                    {
                        mCount++;
                    }
                }
                
            }
            //std::cout<<mCount<<std::endl;
            total.push_back((mCount)/(nCount*(nCount-1.0)));
            std::cout<<x<<": "<<total[x]<<std::endl;
            totalC += total[x];
        }
        std::cout<<totalC<<std::endl;
        double result = ((1.0/n)*totalC);
        std::cout<<"Total: "<<result<<std::endl;
        return 0.0;//result;
}
int main()
{
    std::fstream myfile("toyGraph1.txt",std::ios_base::in);
    int u,v;
    int maxNode = 0;
    std::vector< std::pair<int,int> > allEdges;
    while(myfile >> u >> v)
    {
        allEdges.push_back(std::make_pair(u,v));
        if(u > maxNode)
          maxNode = u;

        if(v > maxNode)
          maxNode = v;                 
    }

    n = maxNode + 1;  //Since nodes starts with 0
    std::cout << "Graph has " << n << " nodes" << std::endl;

    adjMatrix = AdjacencyMatrix(n,std::vector<int>(n,0));
    //populate the matrix
    for(int i =0; i<allEdges.size() ; i++){
       u = allEdges[i].first;
       v = allEdges[i].second;
       adjMatrix[u][v] = 1;
       adjMatrix[v][u] = 1;
    } 
    //You can also make a list of neighbors for each node if you want.
    printAdjMatrix(adjMatrix);


    //TODO: Write serial clustering coefficent code; include timing and error checking
    clustCoeff_Serial(adjMatrix);

    //TODO: Write parallel clustering cofficient code; include timing and error checking
    float d_input[n*n];
    float d_temp[n*n];
    float *d_output;
    float *h_gpu_result = (float*)malloc(n*sizeof(float));
    hipMalloc((void **) &d_input, sizeof(adjMatrix));
    hipMalloc((void **) &d_output, n*sizeof(float));

    for (int i =0;i<n;i++){
	for(int j =0;j<n;j++){
	d_temp[(i*n)+j]=adjMatrix[i][j]; 
	}
    }
    //Source array maps values properly
    for(int i=0;i<n*n;i++)
    {
	printf("%f\n",d_temp[i]);
    }
    //Copying source array to device seems to not be working
    hipMemcpy(d_input, d_temp, (n*n*sizeof(float)), hipMemcpyHostToDevice);
    clustCoeff_Parallel<<<blocks,threads_per_block>>>(d_input,d_output,n);
    hipDeviceSynchronize();
    hipMemcpy(h_gpu_result, d_output, n*sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_output);
    hipFree(d_input);
    for(int j =0;j<n;j++){
    std::cout<<j<<": "<<h_gpu_result[j]<<std::endl;
    }
    //TODO: Compare serial and parallel results

    return 0;
}

