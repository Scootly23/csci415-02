#include "hip/hip_runtime.h"
//CSCI415 - Assignment 2
//Original by: Saeed Salem, 2/25/2015
//Updated by: Otto Borchert, 2/20/2017
//To compile: make clean; make
//To run: ./assign2

#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <math.h>
#include <iomanip>
#include <string>
#include <sys/time.h>

typedef std::vector< std::vector<int> > AdjacencyMatrix;
AdjacencyMatrix adjMatrix;

int threads_per_block = 256;
int blocks = (int)(5000/threads_per_block+1);
int n;

void printAdjMatrix(AdjacencyMatrix adjMatrix)
{
    for (int i=0; i<adjMatrix.size(); i++)
    {
        for (int j=0; j<adjMatrix[i].size(); j++) 
        {
            std::cout << adjMatrix[i][j] << " ";
        }
        std::cout << std::endl;
    }
}
__global__ void clustCoeff_Parallel(int *matrix, float *output,int n)
{
    double totalC = 0.0;
    int x = blockDim.x * blockIdx.x + threadIdx.x;

    if(x < n)
    {
	const int size = (n*n);
	
        int *temp = new int[size];
        int nCount = 0;
        int mCount = 0;
        for(int y = 0;y<n;y++)
        {
	    int b = matrix[x*n+y];
            if(b==1)
            {
                temp[nCount]=y;
                nCount++;
            }

        }
       
        for(int p =0;p<nCount;p++ )
        {
            for(int q =0;q<n;q++)
            {
                if(matrix[temp[p]*n+q] == 1 && matrix[q*n+x] == 1)
                {
                    mCount++;	    
                }
            }
                
        }
        
        output[x]=((mCount)/(nCount*(nCount-1.0)));
        totalC += output[x];
   }
}
double clustCoeff_Serial(AdjacencyMatrix matrix)
{  std::vector<double> total;
        double totalC = 0.0;
        for(int x =0;x<n;x++)
        {
            //Parallelize this hunk
            std::vector<int> temp;
            int nCount = 0;
            int mCount = 0;
            for(int y = 0;y<n;y++)
            {
                if(matrix[x][y])
                {
                    temp.push_back(y);
                    nCount++;
                }

            }
            for(int p =0;p<temp.size();p++ )
            {
                for(int q =0;q<n;q++)
                {
                    if(matrix[temp[p]][q] && matrix[q][x])
                    {
                        mCount++;
                    }
                }
                
            }
            //std::cout<<mCount<<std::endl;
            total.push_back((mCount)/(nCount*(nCount-1.0)));
            totalC += total[x];
        }
        double result = ((1.0/n)*totalC);
        std::cout<<"\nSerial Coeffecient: "<<result<<std::endl;
        return 0.0;//result;
}
long long start_timer() {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	return tv.tv_sec * 1000000 + tv.tv_usec;
}


// Prints the time elapsed since the specified time
long long stop_timer(long long start_time, std::string name) {
	struct timeval tv;
	gettimeofday(&tv, NULL);
	long long end_time = tv.tv_sec * 1000000 + tv.tv_usec;
        std::cout << std::setprecision(5);	
	std::cout << name << ": " << ((float) (end_time - start_time)) / (1000 * 1000) << " sec\n";
	return end_time - start_time;
}
void checkErrors(const char label[])
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}
int main()
{
    std::fstream myfile("toyGraph.txt",std::ios_base::in);
    int u,v;
    int maxNode = 0;
    std::vector< std::pair<int,int> > allEdges;
    while(myfile >> u >> v)
    {
        allEdges.push_back(std::make_pair(u,v));
        if(u > maxNode)
          maxNode = u;

        if(v > maxNode)
          maxNode = v;                 
    }

    n = maxNode + 1;  //Since nodes starts with 0
    std::cout << "Graph has " << n << " nodes" << std::endl;

    adjMatrix = AdjacencyMatrix(n,std::vector<int>(n,0));
    //populate the matrix
    for(int i =0; i<allEdges.size() ; i++){
       u = allEdges[i].first;
       v = allEdges[i].second;
       adjMatrix[u][v] = 1;
       adjMatrix[v][u] = 1;
    } 
    //You can also make a list of neighbors for each node if you want.
    //printAdjMatrix(adjMatrix);


    //TODO: Write serial clustering coefficent code; include timing and error checking
   
    long long serial_start = start_timer();
    clustCoeff_Serial(adjMatrix);
    long long serial_end = stop_timer(serial_start,"Serial Run Time");
    
    //TODO: Write parallel clustering cofficient code; include timing and error checking
    int *d_input;
    //int *d_temp = (int*)malloc(n*n*sizeof(int));
    int *d_temp = new int[n*n];
    float *d_output;
    float *h_gpu_result = (float*)malloc(n*sizeof(float));
    hipMalloc((void **) &d_input, sizeof(int)*n*n);
    hipMalloc((void **) &d_output, n*sizeof(float));
    //checkErrors("MAlloc");
    for (int i =0;i<n;i++){
	for(int j =0;j<n;j++){
	d_temp[(i*n)+j]=adjMatrix[i][j]; 
	}
    }
    hipDeviceSynchronize();
    hipMemcpy(d_input, d_temp, (n*n*sizeof(int)), hipMemcpyHostToDevice);
    //checkErrors("memCopy");

    long long parallel_start = start_timer();
    clustCoeff_Parallel<<<blocks,threads_per_block>>>(d_input,d_output,n);
    hipMemcpy(h_gpu_result, d_output, n*sizeof(float), hipMemcpyDeviceToHost);
    float coef=0.0;
    for(int j =0;j<n;j++){
    coef += h_gpu_result[j];
    }
    
    long long parallel_stop = stop_timer(parallel_start,"\nParallel Run Time");
    std::cout<<"Parallel coeffecient is: "<<(coef/n)<<std::endl;
    //checkErrors("Kernal");
    
    hipDeviceSynchronize();
    hipFree(d_output);
    hipFree(d_input);
    
    //TODO: Compare serial and parallel results

    return 0;
}

